


#include <hip/hip_runtime.h>
#include <stdio.h>



const long int N = 1<<8;
const long int L = N * N;


__global__ void Init(long int n, float * d_a, float heat)
{
    int x = blockIdx.x, y = threadIdx.x;
    int globalIdx = x * blockDim.x + y;

    if(globalIdx < n) { 
        if(x != 0) {
            d_a[globalIdx] = 0;
        }
        else {
            
            if((y == 0) | (y ==  N-1)) {
                d_a[globalIdx] = 0;
            }
            else {
                d_a[globalIdx] = heat;
            }
        }
    }  
}



__global__ void Step(long int n, float * d_old, float * d_new)
{
    int x = blockIdx.x, y = threadIdx.x;
    int globalIdx = x * blockDim.x + y;
    int upper, down, right, left;

    if(globalIdx < n)
    { 
        if((x != 0) && (x !=  N-1))
        {
            if((y != 0) && (y !=  N-1))
            {
                right = (x + 1) * blockDim.x + y;
                left = (x - 1) * blockDim.x + y;
             
                upper = x * blockDim.x + y+1;
                down = x * blockDim.x + y-1;
             
                d_new[globalIdx] = (d_old[upper] + d_old[down] + d_old[right] + d_old[left]) / 4;
                d_old[globalIdx] = d_new[globalIdx];
            }
        }
    }  
}



// CPU code
int main()
{
    float * d_old;
    float * d_new;



    FILE *fp;
    fp = fopen("heatmap.txt","w");

    hipMallocManaged(&d_old, sizeof(float)*L);
    hipMallocManaged(&d_new, sizeof(float)*L);


    Init<<<N,N>>>(L, d_old, 1);
    Init<<<N,N>>>(L, d_new, 1);


    for(int step=0; step < 100000; step++) 

        {
            if (step % 100 == 0) {
                hipDeviceSynchronize();
            }   
         Step<<<N,N>>>(L, d_old, d_new);   
        }
        
        

    

    hipDeviceSynchronize();

    for(int j=0; j<N; j++)
    {
        for (int i=0; i<N; i++)
        {
            fprintf(fp, "%f ", d_new[i + j*N]);
        }
        fprintf(fp, "\n");
    }

    hipDeviceSynchronize();

    hipFree(d_new);
    hipFree(d_old);

    fclose(fp);
    return 0;
}