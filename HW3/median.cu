

#include <hip/hip_runtime.h>
#include <stdio.h>
// kernel

const long int size = 600;
const long int N = size*size;

__global__ void Hello(void)
{

    long int globalIdx = blockDim.x*threadIdx.y+ threadIdx.x;
    //printf("hello from bx = %d\tby = %d\ttx = %d\tty = %d\tgi = %d\n", 
    //    blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, globalIdx);
    globalIdx += 1;
}

__global__ void Init(long int n, float * d_a)
{
    int globalIdx = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    if(globalIdx < n) d_a[globalIdx] = globalIdx;
}





__global__ void Median(long int n, float * d_old, float * d_new, float * d_neigh)
{
    int x = blockIdx.x;
    int y = threadIdx.x;
    int globalIdx = x * blockDim.x + y;
    float stay;

    if(globalIdx < n)
    { 
        if((x != 0) && (x !=  size-1))
        {
            if((y != 0) && (y !=  size-1))
            {
                d_neigh[0] = d_old[globalIdx];
             
                d_neigh[1] = d_old[(x - 1) * blockDim.x + y];
             
                d_neigh[2] = d_old[(x - 1) * blockDim.x + y + 1];
                d_neigh[3] = d_old[x * blockDim.x + y + 1 ];
                d_neigh[4] = d_old[(x + 1) * blockDim.x + y + 1];
             
                d_neigh[5] = d_old[(x + 1) * blockDim.x + y ];
             
                d_neigh[6] = d_old[(x + 1) * blockDim.x + y - 1];
                d_neigh[7] = d_old[x * blockDim.x + y - 1];
                d_neigh[8] = d_old[(x - 1) * blockDim.x + y - 1];
             
                // подумать над сортировкой
                
                for(int i = 0; i < 9; ++i) // i - номер прохода
                {            
                    for(int j = 0; j < 9; ++j) // внутренний цикл прохода
                    {     
                        if (d_neigh[j + 1] < d_neigh[j]) 
                        {
                           stay = d_neigh[j];
                           d_neigh[j ] =  d_neigh[j+1];
                           d_neigh[j+1] = stay;
                        }
                   }
                 }
                
                d_new[globalIdx] = d_neigh[4];
                d_old[globalIdx] = d_new[globalIdx];
            }
        }
    }  
}

// CPU code
int main()
{

    float * h_r = (float *)malloc(sizeof(float)*N);
    float * h_g = (float *)malloc(sizeof(float)*N);
    float * h_b = (float *)malloc(sizeof(float)*N);

    float * d_r;
    float * d_g;
    float * d_b;
    float * d_new;
    float * d_neigh;

    hipMalloc(&d_r, sizeof(float)*N);
    hipMalloc(&d_g, sizeof(float)*N);
    hipMalloc(&d_b, sizeof(float)*N);
    hipMalloc(&d_new, sizeof(float)*N);
    hipMalloc(&d_neigh, sizeof(float)*9);

    FILE *init_r, *init_g, *init_b, *out_r, *out_g, *out_b;
    init_r = fopen("cat_r.txt","r");
    init_g = fopen("cat_g.txt","r");
    init_b = fopen("cat_b.txt","r");

    out_r = fopen("med_cat_r.txt","w");
    out_g = fopen("med_cat_g.txt","w");
    out_b = fopen("med_cat_b.txt","w");

    
    for(int i=0; i<size; i++)
    {
        for(int j=0; j<size; j++)
        {
            fscanf(init_r, "%f", &h_r[i*size + j]);
            fscanf(init_g, "%f", &h_g[i*size + j]);
            fscanf(init_b, "%f", &h_b[i*size + j]);
        }
    }

    hipMemcpy(d_r, h_r, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_g, h_g, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float)*N, hipMemcpyHostToDevice);

    for(int i=0; i<500; i++)
    {
        Median<<<size,size>>>(N, d_r, d_new, d_neigh);
        
        Median<<<size,size>>>(N, d_g, d_new, d_neigh);
        
        Median<<<size,size>>>(N, d_b, d_new, d_neigh);
    }

    hipMemcpy(h_r, d_r, sizeof(float)*N, hipMemcpyDeviceToHost);
    hipMemcpy(h_g, d_g, sizeof(float)*N, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, sizeof(float)*N, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();


    for(int i = 0; i < size; i++)
    {
        for(int j = 0; j < size; j++)
        {
            fprintf(out_r, "%f ", h_r[i*size+j]);
            fprintf(out_g, "%f ", h_g[i*size+j]);
            fprintf(out_b, "%f ", h_b[i*size+j]);
        }
        
        fprintf(out_r, "\n"); 
        fprintf(out_g, "\n"); 
        fprintf(out_b, "\n"); 
    }

    hipDeviceSynchronize();
 
   
    fclose(init_r);
    fclose(init_g);
    fclose(init_b);
    
    fclose(out_r);
    fclose(out_g);
    fclose(out_b);
    
    hipFree(d_r);
    hipFree(d_g);
    hipFree(d_b);
    hipFree(d_new);
    hipFree(d_neigh);
 
    free(h_r);
    free(h_g);
    free(h_b);


    return 0;
}