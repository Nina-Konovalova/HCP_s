

#include <hip/hip_runtime.h>
#include <stdio.h>
// kernel

const long int size = 600;
const long int N = size*size;


__global__ void Init(long int n, float * d_a)
{
    int globalIdx = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    if(globalIdx < n) d_a[globalIdx] = globalIdx;
}


__global__ void Blur(long int n,  float * d_old, float * d_new)
{
    int x = blockIdx.x;
    int y = threadIdx.x;
    int globalIdx = x * blockDim.x + y;
    int up, down, right, left;
    int u_l, d_l, u_r, d_r;

    if(globalIdx < n)
    { 
        if((x != 0) && (x !=  size-1))
        {
            if((y != 0) && (y !=  size-1))
            {
                d_r   = (x + 1) * blockDim.x + y - 1;
                right = (x + 1) * blockDim.x + y;
                u_r   = (x + 1) * blockDim.x + y + 1;
                
                d_l =  (x - 1) * blockDim.x + y - 1;
                left = (x - 1) * blockDim.x + y;
                u_l =  (x - 1) * blockDim.x + y + 1;

                up = x * blockDim.x + y+1;
                down = x * blockDim.x + y-1;
                                
                d_new[globalIdx] = (d_old[globalIdx] + d_old[up] + d_old[down] + d_old[left] +\
                                    d_old[right] + d_old[u_l] + d_old[d_l] + d_old[u_r] + d_old[d_r]) / 9.0;
                d_old[globalIdx] = d_new[globalIdx];
            }
        }
    }  
}



int main()
{
    float * d_r;
    float * d_g;
    float * d_b;
    float * d_new;
   
    hipMalloc(&d_r, sizeof(float)*N);
    hipMalloc(&d_g, sizeof(float)*N);
    hipMalloc(&d_b, sizeof(float)*N);
    hipMalloc(&d_new, sizeof(float)*N);

    float * h_r = (float *)malloc(sizeof(float)*N);
    float * h_g = (float *)malloc(sizeof(float)*N);
    float * h_b = (float *)malloc(sizeof(float)*N);


    FILE *init_r, *init_g, *init_b, *out_r, *out_g, *out_b;
    init_r = fopen("cat_r.txt","r");
    init_g = fopen("cat_g.txt","r");
    init_b = fopen("cat_b.txt","r");

    out_r = fopen("blur_cat_r.txt","w");
    out_g = fopen("blur_cat_g.txt","w");
    out_b = fopen("blur_cat_b.txt","w");


    for(int i = 0; i < size; i++) {
        for(int j = 0; j < size; j++) {
            fscanf(init_r, "%f", &h_r[i*size + j]);
            fscanf(init_g, "%f", &h_g[i*size + j]);
            fscanf(init_b, "%f", &h_b[i*size + j]);
        }
    }

    hipMemcpy(d_r, h_r, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_g, h_g, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float)*N, hipMemcpyHostToDevice);


    for(int i = 0; i < 250; i++) {
        
        Blur<<<size,size>>>(N, d_r, d_new);
 
        Blur<<<size,size>>>(N, d_g, d_new);

        Blur<<<size,size>>>(N, d_b, d_new);

    }

    hipMemcpy(h_r, d_r, sizeof(float)*N, hipMemcpyDeviceToHost);
    hipMemcpy(h_g, d_g, sizeof(float)*N, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, sizeof(float)*N, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    for(int i = 0; i < size; i++) {
        for(int j = 0; j < size; j++) {
            fprintf(out_r, "%f ", h_r[i*size + j]);
            fprintf(out_g, "%f ", h_g[i*size + j]);
            fprintf(out_b, "%f ", h_b[i*size + j]);
        }
        
        fprintf(out_r, "\n"); 
        fprintf(out_g, "\n"); 
        fprintf(out_b, "\n"); 
    }

    hipDeviceSynchronize();
    
    fclose(init_r);
    fclose(init_g);
    fclose(init_b);
    
    fclose(out_r);
    fclose(out_g);
    fclose(out_b);
    
    hipFree(d_r);
    hipFree(d_g);
    hipFree(d_b);
    hipFree(d_new);

    free(h_r);
    free(h_g);
    free(h_b);

    return 0;
}